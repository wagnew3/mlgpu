
#include <hip/hip_runtime.h>
extern "C"

__global__
void sigmoid(float *activation, unsigned int length)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < length; 
         i += blockDim.x * gridDim.x) 
      {
          
          activation[i]=1.0f/(1.0f+__expf(-activation[i]));
          
          //activation[i]=1.0f/(1.0f+expf(-activation[i]));
          
          //activation[i]=activation[i]/(0.5f+0.5f*fabsf(activation[i]))+0.5f;
      }
}

