
#include <hip/hip_runtime.h>
extern "C"

__global__
void sigmoid(float *activation, unsigned int length)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < length; 
         i += blockDim.x * gridDim.x) 
      {
          activation[i]=1.0/(1.0+__expf(-activation[i]));
      }
}

