
#include <hip/hip_runtime.h>
extern "C"

__global__
void crossEntropyCostDerivative(float *desiredOutput, unsigned int length, float *networkOutput, float* result)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < length; 
         i += blockDim.x * gridDim.x) 
      {
          result[i]=-desiredOutput[i]/(0.00001f+networkOutput[i])+(1.0f-desiredOutput[i])/(1.00001f-networkOutput[i]);
      }
}

